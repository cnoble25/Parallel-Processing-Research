#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <limits>
#include <stdio.h>
#include <chrono>


__global__ void sort(int* dev_d, int* dev_c, int col, int row) 
{
    int** con = new int* [row];
    for (int i = 0; i < row; i++) {
        con[i] = new int[col];
    }

    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            con[i][j] = dev_c[i * col + j];
        }
    }
    
    int id = threadIdx.x;
    bool swapped;

    for (int i = 0; i < col - 1; i++) {
        swapped = false;
        for (int j = 0; j < col - i - 1; j++) {
            if (con[id][j] > con[id][j + 1]) {
                int temp = con[id][j];
                con[id][j] = con[id][j + 1];
                con[id][j + 1] = temp;
                swapped = true;
            }
        }
        if (!swapped) {
            break;
        }
    }
    
        for (int j = 0; j < col; j++) {
            dev_d[id * col + j] = con[id][j];
        }
}

hipError_t Parallel_Sort(std::vector<std::vector<int>> * c, int* d)
{
    int row = (*c).size();
    int col = 0;
    //make new_c
    //-------------------------------------------------------
    for (int i = 0; i < (*c).size(); i++) {
        if ((*c)[i].size() > col) {
            col = (*c)[i].size();
        }
    }
    int** new_c = new int* [row];
    for (int i = 0; i < row; i++) {
        new_c[i] = new int[col];
    }
    //-------------------------------------------------------
    //set new c
    //-------------------------------------------------------
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++) {
            if (j >= (*c)[i].size()) {
                new_c[i][j] = INT_MAX;
            }
            else {
                new_c[i][j] = (*c)[i][j];
            }
        }
    }
    //-------------------------------------------------------
    // Flattening Process
    //-------------------------------------------------------
    int * flattened_c = new int[row * col];
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            flattened_c[i * col + j] = new_c[i][j];
        }
    }


    int* dev_c = 0;
    int* dev_d = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);

    cudaStatus = hipMalloc((void**)&dev_c, row * col * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_d, row * col * sizeof(int));

    cudaStatus = hipMemcpy(dev_c, flattened_c, row * col * sizeof(int), hipMemcpyHostToDevice);
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "2 failed!\n");
    }
    sort << <1, row >> > (dev_d, dev_c, col, row);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "3 failed!\n");
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "4 failed!\n");
    }

    cudaStatus = hipMemcpy(d, dev_d, col * row * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "5 failed!\n");
    }
    for (int i = 0; i < (*c).size(); i++) {
        for (int j = 0; j < (*c)[i].size(); j++) {
            (*c)[i][j] = d[col*i + j];
        }
    }


    return cudaStatus;

     
}


int main()
{
    
    const std::vector<int> a = { 1, 2, 3, 4, 5, -5, 10, 20, -42, 78, -1203};
    const std::vector<int> b = { 10, 2, 30, 40, 50, 3, 0 };
    const std::vector<int> e = { 213,213,23,13,5,45,765,3,5,756,3,655,42,1,9,1 };
    std::vector<std::vector<int>> c = {a,b, e};


    int * d = new int[a.size()*b.size()];
    auto start = std::chrono::high_resolution_clock::now();

    hipError_t cudaStatus = Parallel_Sort(&c, d);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
  

    for (int i = 0; i < c.size(); i++) {
        printf("{");
        for (int j = 0; j < c[i].size(); j++) {
            printf("%d, ", c[i][j]);
        }
        printf("}\n");

    }
    auto end = std::chrono::high_resolution_clock::now();

    printf("%d miliseconds", std::chrono::duration_cast<std::chrono::milliseconds>(end - start));
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


    return 0;
}



